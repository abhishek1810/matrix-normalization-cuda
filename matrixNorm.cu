/* Matrix normalization.
 * Compile with "gcc matrixNorm.c"
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <math.h>
#include <sys/types.h>
#include <sys/times.h>
#include <sys/time.h>
#include <time.h>

/* Program Parameters */
#define N 12000  /* Matrix size */

int numBlocks = 32;
int numThreadsPerBlock = 64;

/* Matrices */
volatile float A[N][N], B_cpu[N][N], B_gpu[N][N];

/* Initialize A and B*/
void initialize_inputs() {
    int row, col;
    
    srand((unsigned)time(NULL));
    // srand(0);
    for (row = 0; row < N; row++) {
        for (col = 0; col < N; col++) {
            A[row][col] = (float)rand() / 32768.0;
            B_cpu[row][col] = 0.0;
            B_gpu[row][col] = 0.0;
        }
    }   
}

/* Serial function */
void matrixNormSerially() {
    int row, col;
    float mu, sigma; // Mean and Standard Deviation
    
    printf("Computing Serially.\n");
    
    for (col=0; col < N; col++) {
        mu = 0.0;
        for (row=0; row < N; row++)
            mu += A[row][col];
        mu /= (float) N;
        sigma = 0.0;
        for (row=0; row < N; row++)
            sigma += powf(A[row][col] - mu, 2.0);
        sigma /= (float) N;
        sigma = sqrt(sigma);
        for (row=0; row < N; row++) {
            if (sigma == 0.0)
                B_cpu[row][col] = 0.0;
            else
                B_cpu[row][col] = (A[row][col] - mu) / sigma;
        }
    }
}

/* Method to check correctness of parallel program*/
void checkCorrectness() {
    float sum_cpu = 0; 
    float sum_gpu = 0;
    int row, col;
    for (row=0; row < N; row++) {
        for (col=0; col < N; col++) {
            sum_cpu += B_cpu[row][col];
            sum_gpu += B_gpu[row][col];
        }
    }
    printf("Sum of normalized array by CPU : %f\n", sum_cpu);
    printf("Sum of normalized array by GPU : %f\n", sum_gpu);
}

/* Kernel function */
__global__ void matrixNorm (float *d_A, float *d_B, int n, int totalThreads) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row;
    float mu, sigma;

    for (;col < n; col += totalThreads){
        mu = (float)0.0;
        for (row=0; row < n; row++)
            mu += d_A[row*n+col];
        mu /= (float) n;
        
        // __syncthreads();
        
        sigma = (float)0.0;
        for (row=0; row < n; row++)
            sigma += powf(d_A[row*n+col] - mu, (float)2.0);
        sigma /= (float) n;

        // __syncthreads();
        sigma = sqrt( (float) sigma);

        for (row=0; row < n; row++) {
            if (sigma == (float)0.0)
                d_B[row*n+col] = (float)0.0;
            else
                d_B[row*n+col] = (d_A[row*n+col] - mu) / sigma;
        }
    }
}

/* Print input matrices */
void print_matrix() {
  int row, col;

  if (N <= 5) {
    printf("\nA =\n\t");
    for (row = 0; row < N; row++) {
      for (col = 0; col < N; col++) {
	printf("%5.2f%s", A[row][col], (col < N-1) ? ", " : ";\n\t");
      }
    }
    printf("\nB_cpu = [");
    for (row = 0; row < N; row++) {
      for (col = 0; col < N; col++) {
	    printf("%5.2f%s", B_cpu[row][col], (col < N-1) ? ", " : ";\n\t");
      }
    }
    printf("\nB_gpu = [");
    for (row = 0; row < N; row++) {
      for (col = 0; col < N; col++) {
	    printf("%5.2f%s", B_gpu[row][col], (col < N-1) ? ", " : ";\n\t");
      }
    }
    printf("\n");
  }
}

int main(int argc, char **argv) {
    /* Timing variables */
    struct timeval start, stop;  /* Elapsed times using gettimeofday() */
    struct timezone tzdummy;
    unsigned long long runtime;

    /* Initialize A and B */
    initialize_inputs();

    printf("Computing in Parallel\n");

    float *d_A, *d_B;

    /* Start Clock */
    printf("\n---------------------------------------------\n");
    printf("Matrix size N = %d", N);
    printf("\nStarting clock.\n\n");
    gettimeofday(&start, &tzdummy);

    hipMalloc((void **) &d_A, sizeof(float)*N*N);
    hipMalloc((void **) &d_B, sizeof(float)*N*N);

    hipMemcpy(d_A, (const void *)A, sizeof(float)*N*N, hipMemcpyHostToDevice);

    matrixNorm<<<numBlocks,numThreadsPerBlock>>>(d_A, d_B, N, numBlocks*numThreadsPerBlock);

    hipMemcpy((void *)B_gpu, (d_B), sizeof(float)*N*N, hipMemcpyDeviceToHost);

    /* Stop Clock */
    gettimeofday(&stop, &tzdummy);
    runtime = (unsigned long long)(stop.tv_sec - start.tv_sec) * 1000000 + (stop.tv_usec - start.tv_usec);

    /* Display output */
    // print_B();

    hipFree(d_A);
    hipFree(d_B);

    /* Display timing results */
    printf("Runtime = %g ms.\n", (float)runtime/(float)1000);
    printf("\nStopped clock.");
    printf("\n---------------------------------------------\n");
    
    printf("Computing in Serial\n");
    matrixNormSerially();

    print_matrix();

    printf("Checking correctness\n");
    checkCorrectness();

    exit(0);
}
